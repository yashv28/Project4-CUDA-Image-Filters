#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "bilateralFilter.h"
#include "hip/hip_runtime.h"

#define TIMER 1
namespace bilateralFilter {
	float gaussian[64];
	__constant__ float c_gaussian[64];

	void computeGaussianKernel(float delta, int radius) {
		for (int i = 0; i < 2 * radius + 1; i++) {
			float x = i - radius;
			gaussian[i] = expf(-(x * x) / (2.0f * delta * delta));
		}
	}

	float euclideanLen(float3 a, float3 b, float d) {
		float mod = (b.x - a.x) * (b.x - a.x) +
			(b.y - a.y) * (b.y - a.y) +
			(b.z - a.z) * (b.z - a.z);
		return expf(-mod / (2.0f * d * d));
	}

	void bilateralFilter(float3 *input, float3 *output, float l2norm, size_t cols, size_t rows, int radius) {
		for (int y = 0; y < rows; ++y) {
			for (int x = 0; x < cols; ++x) {
				float sum = 0.0f;
				float3 t = { 0.f, 0.f, 0.f };
				float3 center = input[y * cols + x];
				int r = radius;

				float domainDist = 0.0f, colorDist = 0.0f, factor = 0.0f;

				for (int i = -r; i <= r; i++) {
					int crtY = y + i;
					if (crtY < 0)				crtY = 0;
					else if (crtY >= rows)   	crtY = rows - 1;

					for (int j = -r; j <= r; j++) {
						int crtX = x + j;
						if (crtX < 0) 				crtX = 0;
						else if (crtX >= cols)	 	crtX = cols - 1;

						float3 curPix = input[crtY * cols + crtX];
						domainDist = gaussian[r + i] * gaussian[r + j];
						colorDist = euclideanLen(curPix, center, l2norm);
						factor = domainDist * colorDist;
						sum += factor;
						t = add(t, multiply(factor, curPix));
					}
				}

				output[y * cols + x] = multiply(1.f / sum, t);
			}
		}
	}

	Mat cpufilter(Mat im, float l2norm, int radius)
	{
		im.convertTo(im, CV_32FC3);
		im /= 255;
		size_t rows = im.rows;
		size_t cols = im.cols;
		Mat output(im.size(), im.type());
		float3 * src = (float3*)im.ptr<float3>();
		float3 * dest = (float3*)output.ptr<float3>();
		computeGaussianKernel(l2norm, radius);
#if TIMER
		CpuTimer timer;
		timer.Start();
#endif
		bilateralFilter(src, dest, l2norm, cols, rows, radius);
#if TIMER
		timer.Stop();
		printf("filter: %f s.\n", timer.Elapsed());
#endif
		output *= 255;
		output.convertTo(output, CV_8UC3);
		return output;
	}

	void computeGaussianKernelCuda(float delta, int radius) {
		float h_gaussian[64];
		for (int i = 0; i < 2 * radius + 1; ++i)
		{
			float x = i - radius;
			h_gaussian[i] = expf(-(x * x) / (2.0f * delta * delta));
		}
		hipMemcpyToSymbol(HIP_SYMBOL(c_gaussian), h_gaussian, sizeof(float)*(2 * radius + 1));
	}

	__device__ inline float euclideanLenCuda(float3 a, float3 b, float d) {
		float mod = (b.x - a.x) * (b.x - a.x) + (b.y - a.y) * (b.y - a.y) + (b.z - a.z) * (b.z - a.z);
		return expf(-mod / (2.0f * d * d));
	}

	__device__ inline float3 multiplyCuda(float a, float3 b) {
		return{ a * b.x, a * b.y, a * b.z };
	}

	__device__ inline float3 addCuda(float3 a, float3 b) {
		return{ a.x + b.x, a.y + b.y, a.z + b.z };
	}

	__global__ void bilateralFilterCudaKernel(float3 * dev_input, float3 * dev_output, float l2norm,
		int width, int height, int radius)
	{
		int x = blockIdx.x * blockDim.x + threadIdx.x;
		int y = blockIdx.y * blockDim.y + threadIdx.y;

		if ((x<width) && (y<height))
		{
			float sum = 0.0f;
			float3 t = { 0.f, 0.f, 0.f };
			float3 center = dev_input[y * width + x];
			int r = radius;

			float domainDist = 0.0f, colorDist = 0.0f, factor = 0.0f;

			for (int i = -r; i <= r; i++) {
				int crtY = y + i;
				if (crtY < 0)				crtY = 0;
				else if (crtY >= height)   	crtY = height - 1;

				for (int j = -r; j <= r; ++j) {
					int crtX = x + j;
					if (crtX < 0) 				crtX = 0;
					else if (crtX >= width)	 	crtX = width - 1;

					float3 curPix = dev_input[crtY * width + crtX];
					domainDist = c_gaussian[r + i] * c_gaussian[r + j];
					colorDist = euclideanLenCuda(curPix, center, l2norm);
					factor = domainDist * colorDist;
					sum += factor;
					t = addCuda(t, multiplyCuda(factor, curPix));
				}
			}

			dev_output[y * width + x] = multiplyCuda(1.f / sum, t);
		}
	}

	Mat gpufilter(Mat im, float l2norm, int radius)
	{
		im.convertTo(im, CV_32FC3);
		im /= 255;
		size_t cols = im.cols;
		size_t rows = im.rows;
		Mat result(im.size(), im.type());
		float3 * input = (float3*)im.ptr<float3>();
		float3 * output = (float3*)result.ptr<float3>();
		computeGaussianKernelCuda(l2norm, radius);

		int size = cols * rows * sizeof(float3);
		float3 *dev_input, *dev_output;
		hipMalloc(&dev_input, sizeof(float3)*size);
		hipMalloc(&dev_output, sizeof(float3)*size);
		hipMemcpy(dev_input, input, size, hipMemcpyHostToDevice);

		dim3 block(16, 16);
		dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);
#if TIMER
		GpuTimer timer;
		timer.Start();
#endif
		bilateralFilterCudaKernel << <grid, block >> >(dev_input, dev_output, l2norm, cols, rows, radius);
#if TIMER
		timer.Stop();
		printf("filter kernel: %f msecs.\n", timer.Elapsed());
#endif

		hipDeviceSynchronize();

		hipMemcpy(output, dev_output, size, hipMemcpyDeviceToHost);

		hipFree(dev_input);
		hipFree(dev_output);
		result *= 255;
		result.convertTo(result, CV_8UC3);
		return result;
	}
}